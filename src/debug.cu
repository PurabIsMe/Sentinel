#include "debug.cuh"

void print_array(const char* name, float* array, size_t size, bool is_on_host){
    std::cout << name << ": \n[";
    if(is_on_host){
        for(int i = 0; i < size; i++){
            std::cout << array[i] << ", ";
        }
        std::cout << "]\n";
    }
    else{
        float* dst = (float*)malloc(sizeof(float) * size);
        hipMemcpy(dst,array, sizeof(float)*size, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for(int i = 0; i < size; i++){
            std::cout << dst[i] << ", ";
        }
        std::cout << "]\n";
    }
}

void spawn_device(void** dst, const void* src, size_t size, hipMemcpyKind kind){
    hipMalloc(dst, size);
    hipMemcpy(*dst, src, size, kind);
}