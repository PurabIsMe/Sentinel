#include "hip/hip_runtime.h"
#include "test.cuh"

// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

void add_arrays(float* a, float* b, float* c, int N){
    VecAdd<<<1, N>>>(a, b, c);
    hipDeviceSynchronize();
}

